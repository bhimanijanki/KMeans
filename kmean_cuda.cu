#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

typedef struct
{
	double _r;
	double _g;
	double _b;
	double _m;
	double _n;
	
} Point;

#define CUDA_CALL(x) {if((x) != hipSuccess){ \
  printf("CUDA error at %s:%d\n",__FILE__,__LINE__); \
  printf("  %s\n", hipGetErrorString(hipGetLastError())); \
  exit(EXIT_FAILURE);}} 

void readImageSize(FILE *ifp,int* K,int* a,int* b)
{
fscanf(ifp,"%d\n",K);
printf("%d\n",*K);

fscanf(ifp,"%d\n",a);
printf("%d\n",*a);

fscanf(ifp,"%d\n",b);
printf("%d\n",*b);
}

//reads the ifp file and stores in structure
void readPoints(FILE* ifp,Point *points, int num_points)
{
int i;
for(i=0;i<num_points;i++)
{
fscanf(ifp,"%lf,%lf,%lf,%lf,%lf", &points[i]._r, &points[i]._g, &points[i]._b, &points[i]._m, &points[i]._n);
//printf("%lf,%lf,%lf,%lf,%lf\n", points[i]._r, points[i]._g, points[i]._b, points[i]._m, points[i]._n);
}
}

//Initialize random points as assumed means
void initialize(Point* mean,int K, int num_points, Point* points)
{
int i, a, p=2;
srand(time(NULL));
for(i=0;i<K;i++)
	{
	a = num_points/p;
	//printf("\n num_points: %d\n", num_points/p);
	mean[i]._r = points[a]._r;
	mean[i]._g = points[a]._g;
	mean[i]._b = points[a]._b;
	mean[i]._m = points[a]._m;	
	mean[i]._n = points[a]._n;
	/*mean[i]._r=((double)(rand()%1000))/1000;
	mean[i]._g=((double)(2*rand()%1000))/1000;
	mean[i]._b=((double)(3*rand()%1000))/1000;
	mean[i]._m=((double)(4*rand()%1000))/1000;
	mean[i]._n=((double)(5*rand()%1000))/1000;*/
	//printf("%lf,%lf,%lf,%lf,%lf\n",mean[i]._r,mean[i]._g,mean[i]._b,mean[i]._m,mean[i]._n);
	p++;
	/*mean[i]._r=((double)(rand()%1000))/1000;
	mean[i]._g=((double)(2*rand()%1000))/1000;
	mean[i]._b=((double)(3*rand()%1000))/1000;
	mean[i]._m=((double)(4*rand()%1000))/1000;
	mean[i]._n=((double)(5*rand()%1000))/1000;*/
	}
}

//All points having no clusters
void IntClusterMem(int *cluster, int num_points)
{
int i;
for(i=0;i<num_points;i++)
	{
	cluster[i]=-1;
	}	
}


//to calculate which cluster is the point belonging to.
__global__ void pointsCluster(int* after_cluster_d, Point* point_d,Point* Dmean,int K, int x, int y)
{
	//__shared__ Point Dmean[105];
	//printf("\n%d\t%d\t%d\n",K,x,y);
	int j, k, i;
	j = blockIdx.x*blockDim.x+threadIdx.x;
	k = blockIdx.y*blockDim.y+threadIdx.y;
	//if(j==599 && k==319)
	//printf("%d, %d\n",j,k);
	/*for(i=0;i<K;i++)
	{
		Dmean[i]=mean_d[i];
	}*/
	int parent=0;
	double dist=0;
	int t = (k*(x)+j);
	//if(t>204790)
	//printf("t = %d\n",t);
	double minDist= sqrt((pow((point_d[t]._r-Dmean[0]._r),2)+pow((point_d[t]._g-Dmean[0]._g),2)+pow((point_d[t]._b-Dmean[0]._b),2)+pow((point_d[t]._m-Dmean[0]._m),2)+pow((point_d[t]._n-Dmean[0]._n),2)));
	for(i=1;i<K;i++)
	{	 
		dist = sqrt((pow((point_d[t]._r-Dmean[i]._r),2)+pow((point_d[t]._g-Dmean[i]._g),2)+pow((point_d[t]._b-Dmean[i]._b),2)+pow((point_d[t]._m-Dmean[i]._m),2)+pow((point_d[t]._n-Dmean[i]._n),2)));
			if(minDist>=dist)
			{
				parent=i;
				minDist=dist;
			}
	}
	after_cluster_d[t] = parent;
}


//calculate new mean
void calcNewMean(Point* points,int* cluster,Point* mean,int K,int num_points)
{
Point* newMean=(Point*)malloc(sizeof(Point)*K);
int* members=(int*)malloc(sizeof(int)*(K));
int i;
for(i=0;i<K;i++)
{
	members[i]=0;
	newMean[i]._r=0;
	newMean[i]._g=0;
	newMean[i]._b=0;
	newMean[i]._m=0;
	newMean[i]._n=0;
}	
for(i=0;i<num_points;i++)
{
members[cluster[i]]++;
newMean[cluster[i]]._r+=points[i]._r;
newMean[cluster[i]]._g+=points[i]._g;
newMean[cluster[i]]._b+=points[i]._b;
newMean[cluster[i]]._m+=points[i]._m;
newMean[cluster[i]]._n+=points[i]._n;
}
for(i=0;i<K;i++)
{
if(members[i]!=0.0)
{
newMean[i]._r/=members[i];
newMean[i]._g/=members[i];
newMean[i]._b/=members[i];
newMean[i]._m/=members[i];
newMean[i]._n/=members[i];
}
else
{
newMean[i]._r=0;
newMean[i]._g=0;
newMean[i]._b=0;
newMean[i]._m=0;
newMean[i]._n=0;
}
}
for(i=0;i<K;i++)
{
mean[i]._r=newMean[i]._r;
mean[i]._g=newMean[i]._g;
mean[i]._b=newMean[i]._b;
mean[i]._m=newMean[i]._m;
mean[i]._n=newMean[i]._n;
}	
}

//check for convergence
// it checks that is each points cluster remaining the same
int chkConvrg(int *before_clusters,int *after_cluster,int num_points, float tol)
{
int i;
tol = num_points*tol;
for(i=0;i<num_points;i++)
	{
	if(abs(before_clusters[i]-after_cluster[i])>tol)
		{
		//check = abs(before_clusters[i]-after_cluster[i]);
		//printf("check = %d, after_cluster[%d]=%d, before_clusters[%d]=%d\n",check,i,after_cluster[i],i,before_clusters[i]);
		return -1;
		}
	}
return 0;
}

int main(int argc, char* argv[])
{
//cpu variables
int K;
int num_points;
int * before_clusters;
int i;
int job_done=0;
int x,y,iter=0,iterations;

Point* mean;
Point* points;

int * after_cluster;
float tol;

//gpu variables
Point* points_d;
Point* mean_d;
int * after_cluster_d;
int * before_cluster_d;

hipEvent_t startinit, endinit, startmean, endmean, startcal, endcal, startindex, endindex;
hipEvent_t start1, end1;
float timeinit, timemean, timecal, timeindex;
float time1;
//float totTime = 0;
tol = atof(argv[3]);
//iterations = atof(argv[3]);
//printf("Enter Tolerance:  ");
//scanf("%f",&tol);
printf("Tolerance = %.10f\n",tol);


hipEventCreate(&start1);
hipEventCreate(&end1);
hipEventRecord(start1, 0); 

//Readinf file
FILE *ifp;
ifp=fopen(argv[1],"r");
readImageSize(ifp,&K,&x,&y);
K = atoi(argv[6]);
num_points = x*y;
int blockX=atoi(argv[4]);
int blockY=atoi(argv[5]);
//allocate CPU memory
points=(Point*)malloc(sizeof(Point)*num_points);
readPoints(ifp,points,num_points);
fclose(ifp);
//printf("Input Read Successfully \n");
before_clusters=(int*)malloc(sizeof(int)*num_points);
after_cluster=(int*)malloc(sizeof(int)*num_points);
mean=(Point*)malloc(sizeof(Point)*K);

//initializing to default values
initialize(mean,K, num_points, points);
IntClusterMem(before_clusters,num_points);
IntClusterMem(after_cluster,num_points);


//printf("points = %lf",points[0]._r);
//allocate gpu memory
//printf("No problem till Here1\n");
CUDA_CALL(hipMalloc((void**) &after_cluster_d, sizeof(int)*num_points));
CUDA_CALL(hipMalloc((void**) &before_cluster_d, sizeof(int)*num_points));
CUDA_CALL(hipMalloc((void**) &points_d, sizeof(Point)*num_points));
CUDA_CALL(hipMalloc((void**) &mean_d, sizeof(Point)*K));
//printf("No problem till Here2\n");

hipEventCreate(&startinit);
hipEventCreate(&endinit);
hipEventRecord(startinit, 0); 

//copy data points to device
CUDA_CALL(hipMemcpy(points_d, points, sizeof(Point)*num_points, hipMemcpyHostToDevice));
CUDA_CALL(hipMemcpy(after_cluster_d, after_cluster, sizeof(int)*num_points, hipMemcpyHostToDevice));

hipEventRecord(endinit, 0);
hipEventSynchronize(endinit);
hipEventElapsedTime(&timeinit, startinit, endinit);
//printf("No problem till Here3\n");


while(1)
{	
	//printf("No problem till Here4\n");
	iter++;
	hipEventCreate(&startmean);
	hipEventCreate(&endmean);
	hipEventRecord(startmean, 0); 
	//copy initial centroids to device
	CUDA_CALL(hipMemcpy(mean_d, mean, sizeof(Point)*K, hipMemcpyHostToDevice));
	hipEventRecord(endmean, 0);
	hipEventSynchronize(endmean);
	hipEventElapsedTime(&timemean, startmean, endmean);	
	//cuda memory copy
	//CUDA_CALL(cudaMemcpy(after_cluster_d, after_cluster, sizeof(int)*num_points, cudaMemcpyHostToDevice));
	//CUDA_CALL(cudaMemcpy(before_cluster_d, before_clusters, sizeof(int)*num_points, cudaMemcpyHostToDevice));
	//CUDA_CALL(cudaMemcpy(x_d, &x, sizeof(int), cudaMemcpyHostToDevice));
	//CUDA_CALL(cudaMemcpy(y_d, &y, sizeof(int), cudaMemcpyHostToDevice));
	//CUDA_CALL(cudaMemcpy(K_d, &K, sizeof(int), cudaMemcpyHostToDevice));
	hipEventCreate(&startcal);
	hipEventCreate(&endcal);
	hipEventRecord(startcal, 0); 

	dim3 block(blockX, blockY);
	dim3 grid((x+blockX-1)/blockX, (y+blockY-1)/blockY);

	pointsCluster<<<grid,block>>>(after_cluster_d, points_d,mean_d,K,x,y);

	//printf("Time taken by parallel portion: %f\n",time);
	//totTime +=time;
	//printf("No problem till Here5\n");
	hipDeviceSynchronize();	 
	hipEventRecord(endcal, 0);
	hipEventSynchronize(endcal);
	hipEventElapsedTime(&timecal, startcal, endcal);

	hipEventCreate(&startindex);
	hipEventCreate(&endindex);
	hipEventRecord(startindex, 0); 
	CUDA_CALL(hipMemcpy(after_cluster, after_cluster_d, sizeof(int)*num_points, hipMemcpyDeviceToHost));
	hipEventRecord(endindex, 0);
	hipEventSynchronize(endindex);
	hipEventElapsedTime(&timeindex, startindex, endindex);	
	calcNewMean(points,after_cluster,mean,K,num_points);
	//printf("New Centroids are calculated!\n");

	if(iter>iterations)
	{
		printf("K-mean algorithm Converged with iterations = %d!\n",iter);
		job_done=1;
		
	}
	else
	{
		//printf("Not converged!\n");
		for(i=0;i<num_points;i++)
		{
			//printf("1 after_cluster[%d]=%d, before_clusters[%d]=%d\n",i,after_cluster[i],i,before_clusters[i]);
			
			before_clusters[i]=after_cluster[i];
			
			//printf("after_cluster[%d]=%d, before_clusters[%d]=%d\n",i,after_cluster[i],i,before_clusters[i]);
		}

		
	}
	
	if(job_done==1)
		break;

}

	

//Outputting to the ofp file
FILE* ofp=fopen(argv[2],"w");
fprintf(ofp,"%d\n",K);
fprintf(ofp,"%d\n",x);
fprintf(ofp,"%d\n",y);
for(i=0;i<K;i++)
fprintf(ofp,"%lf,%lf,%lf,%lf,%lf\n",mean[i]._r,mean[i]._g,mean[i]._b,mean[i]._m,mean[i]._n);
for(i=0;i<num_points;i++)
fprintf(ofp,"%lf,%lf,%lf,%lf,%lf,%d\n",points[i]._r,points[i]._g,points[i]._b,points[i]._m,points[i]._n,after_cluster[i]+1);
fclose(ofp);
hipEventRecord(end1, 0);
hipEventSynchronize(end1);
hipEventElapsedTime(&time1, start1, end1);
printf("Time for sending initial data from host to device : %f\t sec\n",timeinit/1000);
printf("Time for sending new means from host to device : %f\t sec\n",timemean/1000);
printf("Time for calculation : %f\t sec\n",timecal/1000);
printf("Time for sending new index from device to host : %f\t sec\n",timeindex/1000);	
printf("Total Time : %f\t sec\n",time1/1000);
CUDA_CALL(hipFree(after_cluster_d));
CUDA_CALL(hipFree(mean_d));
CUDA_CALL(hipFree(points_d));
free(before_clusters);
free(mean);
free(points);
free(after_cluster);


//End of all
     return 0;
}

